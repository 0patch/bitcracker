#include "hip/hip_runtime.h"
/*
 * BitCracker: BitLocker password cracking tool, CUDA version.
 * Copyright (C) 2013-2017  Elena Ago <elena dot ago at gmail dot com>
 *							Massimo Bernaschi <massimo dot bernaschi at gmail dot com>
 * 
 * This file is part of BitCracker.
 * 
 * BitCracker is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 * 
 * BitCracker is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with BitCracker. If not, see <http://www.gnu.org/licenses/>.
 */

#include "bitcracker.h"

texture<uint32_t> w_texture;
texture<uint8_t> w_password0;
texture<uint8_t> w_password1;
int 	*deviceFound[2], *hostFound[2];
char	*hostPassword[2], *devicePassword[2];
char 	outPsw[MAX_INPUT_PASSWORD_LEN+1];
int 	outIndexPsw=0;

static int check_match(int iStream) {
	int i=0;

	if (*hostFound[iStream] >= 0){
		outIndexPsw=*(hostFound[iStream]);
		snprintf(outPsw, MAX_INPUT_PASSWORD_LEN+1, (char *)(hostPassword[iStream]+(outIndexPsw*FIXED_PASSWORD_BUFFER)));
		for(i=0; i<MAX_INPUT_PASSWORD_LEN; i++)
			if(outPsw[i] == 0x80 || outPsw[i] == 0xffffff80) outPsw[i]='\0';

		return 1;
	}

	return 0;
}
char *cuda_attack(char *dname, uint32_t * w_blocks_d, unsigned char * encryptedVMK, unsigned char * nonce,  int gridBlocks) {

	FILE		*fp;
	int			indexStream, numReadPassword[2], firstLoop, match=0;
	long long	totPsw = 0;

	uint8_t		tmpIV[IV_SIZE], *deviceEncryptedVMK, *deviceIV;


	hipEvent_t	start[2], stop[2];
	hipStream_t stream[2];
	float elapsedTime;

	if(dname == NULL || w_blocks_d == NULL || encryptedVMK == NULL)
	{
		fprintf(stderr, "Attack input error\n");
		return NULL;
	}

	if(tot_psw <= 0)
	{
		fprintf(stderr, "Attack tot passwords error: %d\n", tot_psw);
		return NULL;
	}
	
	//-------- IV setup ------
	memset(tmpIV, 0, IV_SIZE);
	memcpy(tmpIV + 1, nonce, NONCE_SIZE);
	if(IV_SIZE-1 - NONCE_SIZE - 1 < 0)
	{
		fprintf(stderr, "Attack nonce error\n");
		return NULL;
	}
	*tmpIV = (unsigned char)(IV_SIZE - 1 - NONCE_SIZE - 1);
	tmpIV[IV_SIZE-1] = 1; 
	// -----------------------

	// ---- Open File Dictionary ----
	if (!memcmp(dname, "-\0", 2)) {
		fp = stdin;
	} else {
		fp = fopen(dname, "r");
		if (!fp) {
			fprintf(stderr, "Can't open dictionary file %s.\n", dname);
			return NULL;
		}
	}
	// -------------------------------

	// ---- HOST VARIABLES ----
	BITCRACKER_CUDA_CHECK( hipHostAlloc( (void ** ) &hostPassword[0], size_psw, hipHostMallocDefault) );
	BITCRACKER_CUDA_CHECK( hipHostAlloc( (void ** ) &hostPassword[1], size_psw, hipHostMallocDefault) );
	BITCRACKER_CUDA_CHECK( hipHostAlloc( (void ** ) &hostFound[0], sizeof(uint32_t), hipHostMallocDefault) );
	BITCRACKER_CUDA_CHECK( hipHostAlloc( (void ** ) &hostFound[1], sizeof(uint32_t), hipHostMallocDefault) );
	*hostFound[0] = *hostFound[1] = -1;
	// ------------------------

	// ---- CUDA VARIABLES ----
	BITCRACKER_CUDA_CHECK( hipMalloc( (void ** ) &deviceEncryptedVMK, VMK_DECRYPT_SIZE*sizeof(uint8_t)) );
	BITCRACKER_CUDA_CHECK( hipMemcpy(deviceEncryptedVMK, encryptedVMK, VMK_DECRYPT_SIZE*sizeof(uint8_t), hipMemcpyHostToDevice) );
	
	BITCRACKER_CUDA_CHECK( hipMalloc( (void ** ) &deviceIV, IV_SIZE*sizeof(uint8_t)) );
	BITCRACKER_CUDA_CHECK( hipMemcpy(deviceIV, tmpIV, IV_SIZE*sizeof(uint8_t), hipMemcpyHostToDevice) );

	BITCRACKER_CUDA_CHECK( hipMalloc( (void ** ) &devicePassword[0], (size_psw * sizeof(uint8_t)) ) );
	BITCRACKER_CUDA_CHECK( hipMalloc( (void ** ) &devicePassword[1], (size_psw * sizeof(uint8_t)) ) );

	BITCRACKER_CUDA_CHECK( hipMalloc( (void ** ) &deviceFound[0], (sizeof(uint32_t)) ) );
	BITCRACKER_CUDA_CHECK( hipMalloc( (void ** ) &deviceFound[1], (sizeof(uint32_t)) ) );
	
	BITCRACKER_CUDA_CHECK( hipMemcpy(deviceFound[0], hostFound[0], sizeof(uint32_t), hipMemcpyHostToDevice) );
	BITCRACKER_CUDA_CHECK( hipMemcpy(deviceFound[1], hostFound[1], sizeof(uint32_t), hipMemcpyHostToDevice) );

	BITCRACKER_CUDA_CHECK( hipStreamCreate(&(stream[0])) );
	BITCRACKER_CUDA_CHECK( hipStreamCreate(&(stream[1])) );

	BITCRACKER_CUDA_CHECK( hipEventCreate(&start[0]) );
	BITCRACKER_CUDA_CHECK( hipEventCreate(&start[1]) );
	BITCRACKER_CUDA_CHECK( hipEventCreate(&stop[0]) );
	BITCRACKER_CUDA_CHECK( hipEventCreate(&stop[1]) );
	// ---------------------

	// -------- TEXTURE --------
	BITCRACKER_CUDA_CHECK(hipBindTexture(NULL, w_texture, w_blocks_d, (SINGLE_BLOCK_SHA_SIZE * ITERATION_NUMBER * sizeof(uint32_t))));
	BITCRACKER_CUDA_CHECK(hipBindTexture(NULL, w_password0, devicePassword[0], (size_psw * sizeof(uint8_t))));
	BITCRACKER_CUDA_CHECK(hipBindTexture(NULL, w_password1, devicePassword[1], (size_psw * sizeof(uint8_t))));
	// -------------------------

	//	BITCRACKER_CUDA_CHECK (hipDeviceSetCacheConfig( hipFuncCachePreferL1 ) );

	printf("Starting CUDA attack:\n\tCUDA Threads: %d\n\tCUDA Blocks: %d\n\tPsw per thread: %d\n\tMax Psw per kernel: %d\n\tDictionary: %s\n\n", 
		ATTACK_DEFAULT_THREADS, gridBlocks, psw_x_thread, tot_psw, (fp == stdin)?"standard input":dname);

	indexStream = 1;
	firstLoop=TRUE;
	while(!feof(fp)) {
		indexStream ^= 1;
		numReadPassword[indexStream] = readFilePassword(&hostPassword[indexStream], tot_psw, fp);
	
		BITCRACKER_CUDA_CHECK( hipMemcpyAsync(devicePassword[indexStream], hostPassword[indexStream], size_psw, hipMemcpyHostToDevice, stream[indexStream]) );
		
		if(firstLoop == FALSE)
			BITCRACKER_CUDA_CHECK( hipStreamSynchronize(stream[indexStream^1]) );
	
		BITCRACKER_CUDA_CHECK( hipEventRecord(start[indexStream], stream[indexStream]) );
		decrypt_vmk<<<gridBlocks, ATTACK_DEFAULT_THREADS, 0, stream[indexStream]>>>(indexStream, numReadPassword[indexStream], deviceFound[indexStream], deviceEncryptedVMK, deviceIV);
		BITCRACKER_CUDA_CHECK_LAST_ERROR();
		BITCRACKER_CUDA_CHECK( hipEventRecord(stop[indexStream], stream[indexStream]) );		
		BITCRACKER_CUDA_CHECK( hipMemcpyAsync(hostFound[indexStream], deviceFound[indexStream], sizeof(unsigned int), hipMemcpyDeviceToHost, stream[indexStream]) );
	
		if(firstLoop == FALSE)
		{
			totPsw += numReadPassword[indexStream^1];
			BITCRACKER_CUDA_CHECK( hipEventElapsedTime(&elapsedTime, start[indexStream^1], stop[indexStream^1]) );
			
			printf("CUDA Kernel execution:\n\tStream %d\n\tEffective number psw: %d\n\tTime: %f sec\n\tPasswords x second: %8.2f pw/sec\n", 
							indexStream^1, numReadPassword[indexStream^1], ATTACK_DEFAULT_THREADS, gridBlocks, (elapsedTime/1000.0), numReadPassword[indexStream^1]/(elapsedTime/1000.0));
			
			match=check_match(indexStream^1);
			if(match) break;
		}

    	firstLoop = FALSE;
	}

	BITCRACKER_CUDA_CHECK( hipStreamSynchronize(stream[indexStream]) );
	
	if (fp != stdin)
		fclose(fp);

	if (*hostFound[indexStream^1] < 0) {
		totPsw += numReadPassword[indexStream];
		BITCRACKER_CUDA_CHECK( hipEventElapsedTime(&elapsedTime, start[indexStream], stop[indexStream]) );
		printf("CUDA Kernel execution:\n\tStream %d\n\tEffective number psw: %d\n\tTime: %f sec\n\tPasswords x second: %8.2f pw/sec\n", 
			indexStream, numReadPassword[indexStream], (elapsedTime/1000.0), numReadPassword[indexStream]/(elapsedTime/1000.0));

		match=check_match(indexStream);
	}

	if(match==1)
		printf("\n\n================================================\nCUDA attack completed\nPasswords evaluated: %d\nPassword found: [%s]\n================================================\n\n", totPsw, outPsw);
	else
		printf("\n\n================================================\nCUDA attack completed\nPasswords evaluated: %d\nPassword not found!\n================================================\n\n", totPsw);

	BITCRACKER_CUDA_CHECK( hipUnbindTexture(&w_password0) );
	BITCRACKER_CUDA_CHECK( hipUnbindTexture(&w_password1) );

	BITCRACKER_CUDA_CHECK( hipHostFree(hostPassword[0]) );
	BITCRACKER_CUDA_CHECK( hipHostFree(hostPassword[1]) );
	BITCRACKER_CUDA_CHECK( hipFree(devicePassword[0]) );
	BITCRACKER_CUDA_CHECK( hipFree(devicePassword[1]) );
	BITCRACKER_CUDA_CHECK( hipFree(deviceFound[0]) );
	BITCRACKER_CUDA_CHECK( hipFree(deviceFound[1]) );
	BITCRACKER_CUDA_CHECK( hipStreamDestroy(stream[0]) );
	BITCRACKER_CUDA_CHECK( hipStreamDestroy(stream[1]) );
	BITCRACKER_CUDA_CHECK( hipUnbindTexture(&w_texture) );
	

	return NULL;
}


#define END_STRING 0x80 //0xFF
//16 byte per password + 1 byte per length
__global__ void decrypt_vmk(int numStream, int tot_psw_kernel, int *found, unsigned char * vmkKey, unsigned char * IV) {
    int globalIndexPassword = (threadIdx.x+blockIdx.x*blockDim.x);
	
	//Avoid register spilling in local memory
	uint32_t hash0;
	uint32_t hash1;
	uint32_t hash2;
	uint32_t hash3;
	uint32_t hash4;
	uint32_t hash5;
	uint32_t hash6;
	uint32_t hash7;

	uint32_t schedule0;
	uint32_t schedule1;
	uint32_t schedule2;
	uint32_t schedule3;
	uint32_t schedule4;
	uint32_t schedule5;
	uint32_t schedule6;
	uint32_t schedule7;
	uint32_t schedule8;
	uint32_t schedule9;
	uint32_t schedule10;
	uint32_t schedule11;
	uint32_t schedule12;
	uint32_t schedule13;
	uint32_t schedule14;
	uint32_t schedule15;
	uint32_t schedule16;
	uint32_t schedule17;
	uint32_t schedule18;
	uint32_t schedule19;
	uint32_t schedule20;
	uint32_t schedule21;
	uint32_t schedule22;
	uint32_t schedule23;
	uint32_t schedule24;
	uint32_t schedule25;
	uint32_t schedule26;
	uint32_t schedule27;
	uint32_t schedule28;
	uint32_t schedule29;
	uint32_t schedule30;
	uint32_t schedule31;

	uint32_t a,b,c,d,e,f,g,h;
	int index_generic;
	uint32_t first_hash0;
	uint32_t first_hash1;
	uint32_t first_hash2;
	uint32_t first_hash3;
	uint32_t first_hash4;
	uint32_t first_hash5;
	uint32_t first_hash6;
	uint32_t first_hash7;

	uint32_t indexW=(globalIndexPassword*FIXED_PASSWORD_BUFFER);
	int8_t curr_fetch=0;
	//int8_t stop=0;

	while(globalIndexPassword < tot_psw_kernel)
	{
		
		first_hash0 = UINT32_C(0x6A09E667);
		first_hash1 = UINT32_C(0xBB67AE85);
		first_hash2 = UINT32_C(0x3C6EF372);
		first_hash3 = UINT32_C(0xA54FF53A);
		first_hash4 = UINT32_C(0x510E527F);
		first_hash5 = UINT32_C(0x9B05688C);
		first_hash6 = UINT32_C(0x1F83D9AB);
		first_hash7 = UINT32_C(0x5BE0CD19);

		a = UINT32_C(0x6A09E667);
		b = UINT32_C(0xBB67AE85);
		c = UINT32_C(0x3C6EF372);
		d = UINT32_C(0xA54FF53A);
		e = UINT32_C(0x510E527F);
		f = UINT32_C(0x9B05688C);
		g = UINT32_C(0x1F83D9AB);
		h = UINT32_C(0x5BE0CD19);

//----------------------------------------------------- FIRST HASH ------------------------------------------------
		indexW=(globalIndexPassword*FIXED_PASSWORD_BUFFER);
		curr_fetch=0;
		index_generic=MAX_INPUT_PASSWORD_LEN;
		//stop=0;
		if(numStream == 0)
		{
			schedule0 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;
			schedule1 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;
			schedule2 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;
			schedule3 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;

			schedule4 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule5 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch; /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1; /* stop=1; */ }
			curr_fetch+=2;

			schedule6 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule7 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule8 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule9 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule10 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule11 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule12 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			//27
			schedule13 = ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password0, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password0, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password0, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			//curr_fetch+=2;
		}
		else
		{
			schedule0 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;
			schedule1 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;
			schedule2 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;
			schedule3 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			curr_fetch+=2;

			schedule4 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule5 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule6 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule7 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule8 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule9 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule10 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule11 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			schedule12 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			curr_fetch+=2;

			//27
			schedule13 = ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch)) << 24) | 0 | ((uint32_t)tex1Dfetch(w_password1, (indexW+curr_fetch+1)) <<  8) | 0;
			if(tex1Dfetch(w_password1, (indexW+curr_fetch)) == END_STRING) { index_generic=curr_fetch;  /* stop=1; */ }
			if(tex1Dfetch(w_password1, (indexW+curr_fetch+1)) == END_STRING) { index_generic=curr_fetch+1;  /* stop=1; */ }
			//curr_fetch+=2;		
		}


		if(index_generic == MAX_INPUT_PASSWORD_LEN) schedule13 = schedule13 | ((uint32_t)0x8000);
		//64-bit
		schedule14=0;
		index_generic*=2;
		schedule15 = ((uint8_t)((index_generic << 3) >> 8)) << 8 | ((uint8_t)(index_generic << 3));


		printf("thread: %d, indexW: %d, index_generic %d, MAX_INPUT_PASSWORD_LEN %d\n", globalIndexPassword, indexW, index_generic, MAX_INPUT_PASSWORD_LEN);
		printf("thread: %d schedule0: %x\n", globalIndexPassword, schedule0);
		printf("thread: %d schedule1: %x\n", globalIndexPassword, schedule1);
		printf("thread: %d schedule2: %x\n", globalIndexPassword, schedule2);
		printf("thread: %d schedule3: %x\n", globalIndexPassword, schedule3);
		printf("thread: %d schedule4: %x\n", globalIndexPassword, schedule4);
		printf("thread: %d schedule5: %x\n", globalIndexPassword, schedule5);
		printf("thread: %d schedule6: %x\n", globalIndexPassword, schedule6);
		printf("thread: %d schedule7: %x\n", globalIndexPassword, schedule7);
		printf("thread: %d schedule8: %x\n", globalIndexPassword, schedule8);
		printf("thread: %d schedule9: %x\n", globalIndexPassword, schedule9);
		printf("thread: %d schedule10: %x\n", globalIndexPassword, schedule10);
		printf("thread: %d schedule11: %x\n", globalIndexPassword, schedule11);
		printf("thread: %d schedule12: %x\n", globalIndexPassword, schedule12);
		printf("thread: %d schedule13: %x\n", globalIndexPassword, schedule13);
		printf("thread: %d schedule14: %x\n", globalIndexPassword, schedule14);
		printf("thread: %d schedule15: %x\n", globalIndexPassword, schedule15);

		ALL_SCHEDULE_LAST16()

		ROUND(a, b, c, d, e, f, g, h,  schedule0, 0x428A2F98)
		ROUND(h, a, b, c, d, e, f, g,  schedule1, 0x71374491)
		ROUND(g, h, a, b, c, d, e, f,  schedule2, 0xB5C0FBCF)
		ROUND(f, g, h, a, b, c, d, e,  schedule3, 0xE9B5DBA5)
		ROUND(e, f, g, h, a, b, c, d,  schedule4, 0x3956C25B)
		ROUND(d, e, f, g, h, a, b, c,  schedule5, 0x59F111F1)
		ROUND(c, d, e, f, g, h, a, b,  schedule6, 0x923F82A4)
		ROUND(b, c, d, e, f, g, h, a,  schedule7, 0xAB1C5ED5)
		ROUND(a, b, c, d, e, f, g, h,  schedule8, 0xD807AA98)
		ROUND(h, a, b, c, d, e, f, g,  schedule9, 0x12835B01)
		ROUND(g, h, a, b, c, d, e, f, schedule10, 0x243185BE)
		ROUND(f, g, h, a, b, c, d, e, schedule11, 0x550C7DC3)
		ROUND(e, f, g, h, a, b, c, d, schedule12, 0x72BE5D74)
		ROUND(d, e, f, g, h, a, b, c, schedule13, 0x80DEB1FE)
		ROUND(c, d, e, f, g, h, a, b, schedule14, 0x9BDC06A7)
		ROUND(b, c, d, e, f, g, h, a, schedule15, 0xC19BF174)
		ROUND(a, b, c, d, e, f, g, h, schedule16, 0xE49B69C1)
		ROUND(h, a, b, c, d, e, f, g, schedule17, 0xEFBE4786)
		ROUND(g, h, a, b, c, d, e, f, schedule18, 0x0FC19DC6)
		ROUND(f, g, h, a, b, c, d, e, schedule19, 0x240CA1CC)
		ROUND(e, f, g, h, a, b, c, d, schedule20, 0x2DE92C6F)
		ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4A7484AA)
		ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5CB0A9DC)
		ROUND(b, c, d, e, f, g, h, a, schedule23, 0x76F988DA)
		ROUND(a, b, c, d, e, f, g, h, schedule24, 0x983E5152)
		ROUND(h, a, b, c, d, e, f, g, schedule25, 0xA831C66D)
		ROUND(g, h, a, b, c, d, e, f, schedule26, 0xB00327C8)
		ROUND(f, g, h, a, b, c, d, e, schedule27, 0xBF597FC7)
		ROUND(e, f, g, h, a, b, c, d, schedule28, 0xC6E00BF3)
		ROUND(d, e, f, g, h, a, b, c, schedule29, 0xD5A79147)
		ROUND(c, d, e, f, g, h, a, b, schedule30, 0x06CA6351)
		ROUND(b, c, d, e, f, g, h, a, schedule31, 0x14292967)

		ALL_SCHEDULE32()

		ROUND(a, b, c, d, e, f, g, h, schedule0, 0x27B70A85)
		ROUND(h, a, b, c, d, e, f, g, schedule1, 0x2E1B2138)
		ROUND(g, h, a, b, c, d, e, f, schedule2, 0x4D2C6DFC)
		ROUND(f, g, h, a, b, c, d, e, schedule3, 0x53380D13)
		ROUND(e, f, g, h, a, b, c, d, schedule4, 0x650A7354)
		ROUND(d, e, f, g, h, a, b, c, schedule5, 0x766A0ABB)
		ROUND(c, d, e, f, g, h, a, b, schedule6, 0x81C2C92E)
		ROUND(b, c, d, e, f, g, h, a, schedule7, 0x92722C85)
		ROUND(a, b, c, d, e, f, g, h, schedule8, 0xA2BFE8A1)
		ROUND(h, a, b, c, d, e, f, g, schedule9, 0xA81A664B)
		ROUND(g, h, a, b, c, d, e, f, schedule10, 0xC24B8B70)
		ROUND(f, g, h, a, b, c, d, e, schedule11, 0xC76C51A3)
		ROUND(e, f, g, h, a, b, c, d, schedule12, 0xD192E819)
		ROUND(d, e, f, g, h, a, b, c, schedule13, 0xD6990624)
		ROUND(c, d, e, f, g, h, a, b, schedule14, 0xF40E3585)
		ROUND(b, c, d, e, f, g, h, a, schedule15, 0x106AA070)
		ROUND(a, b, c, d, e, f, g, h, schedule16, 0x19A4C116)
		ROUND(h, a, b, c, d, e, f, g, schedule17, 0x1E376C08)
		ROUND(g, h, a, b, c, d, e, f, schedule18, 0x2748774C)
		ROUND(f, g, h, a, b, c, d, e, schedule19, 0x34B0BCB5)
		ROUND(e, f, g, h, a, b, c, d, schedule20, 0x391C0CB3)
		ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4ED8AA4A)
		ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5B9CCA4F)
		ROUND(b, c, d, e, f, g, h, a, schedule23, 0x682E6FF3)
		ROUND(a, b, c, d, e, f, g, h, schedule24, 0x748F82EE)
		ROUND(h, a, b, c, d, e, f, g, schedule25, 0x78A5636F)
		ROUND(g, h, a, b, c, d, e, f, schedule26, 0x84C87814)
		ROUND(f, g, h, a, b, c, d, e, schedule27, 0x8CC70208)
		ROUND(e, f, g, h, a, b, c, d, schedule28, 0x90BEFFFA)
		ROUND(d, e, f, g, h, a, b, c, schedule29, 0xA4506CEB)
		ROUND(c, d, e, f, g, h, a, b, schedule30, 0xBEF9A3F7)
		ROUND(b, c, d, e, f, g, h, a, schedule31, 0xC67178F2)
		
		first_hash0 += a;
		first_hash1 += b;
		first_hash2 += c;
		first_hash3 += d;
		first_hash4 += e;
		first_hash5 += f;
		first_hash6 += g;
		first_hash7 += h;



//----------------------------------------------------- SECOND HASH ------------------------------------------------
		//old loadschedule
		schedule0 = first_hash0;
		schedule1 = first_hash1;
		schedule2 = first_hash2;
		schedule3 = first_hash3;
		schedule4 = first_hash4;
		schedule5 = first_hash5;
		schedule6 = first_hash6;
		schedule7 = first_hash7;
		schedule8 = 0x80000000;
		schedule9 = 0;
		schedule10 = 0;
		schedule11 = 0;
		schedule12 = 0;
		schedule13 = 0;
		schedule14 = 0;
		schedule15 = 0x100;

		first_hash0 = UINT32_C(0x6A09E667);
		first_hash1 = UINT32_C(0xBB67AE85);
		first_hash2 = UINT32_C(0x3C6EF372);
		first_hash3 = UINT32_C(0xA54FF53A);
		first_hash4 = UINT32_C(0x510E527F);
		first_hash5 = UINT32_C(0x9B05688C);
		first_hash6 = UINT32_C(0x1F83D9AB);
		first_hash7 = UINT32_C(0x5BE0CD19);

		a = first_hash0;
		b = first_hash1;
		c = first_hash2;
		d = first_hash3;
		e = first_hash4;
		f = first_hash5;
		g = first_hash6;
		h = first_hash7;

		ALL_SCHEDULE_LAST16()

		ROUND(a, b, c, d, e, f, g, h,  schedule0, 0x428A2F98)
		ROUND(h, a, b, c, d, e, f, g,  schedule1, 0x71374491)
		ROUND(g, h, a, b, c, d, e, f,  schedule2, 0xB5C0FBCF)
		ROUND(f, g, h, a, b, c, d, e,  schedule3, 0xE9B5DBA5)
		ROUND(e, f, g, h, a, b, c, d,  schedule4, 0x3956C25B)
		ROUND(d, e, f, g, h, a, b, c,  schedule5, 0x59F111F1)
		ROUND(c, d, e, f, g, h, a, b,  schedule6, 0x923F82A4)
		ROUND(b, c, d, e, f, g, h, a,  schedule7, 0xAB1C5ED5)
		ROUND(a, b, c, d, e, f, g, h,  schedule8, 0xD807AA98)
		ROUND(h, a, b, c, d, e, f, g,  schedule9, 0x12835B01)
		ROUND(g, h, a, b, c, d, e, f, schedule10, 0x243185BE)
		ROUND(f, g, h, a, b, c, d, e, schedule11, 0x550C7DC3)
		ROUND(e, f, g, h, a, b, c, d, schedule12, 0x72BE5D74)
		ROUND(d, e, f, g, h, a, b, c, schedule13, 0x80DEB1FE)
		ROUND(c, d, e, f, g, h, a, b, schedule14, 0x9BDC06A7)
		ROUND(b, c, d, e, f, g, h, a, schedule15, 0xC19BF174)
		ROUND(a, b, c, d, e, f, g, h, schedule16, 0xE49B69C1)
		ROUND(h, a, b, c, d, e, f, g, schedule17, 0xEFBE4786)
		ROUND(g, h, a, b, c, d, e, f, schedule18, 0x0FC19DC6)
		ROUND(f, g, h, a, b, c, d, e, schedule19, 0x240CA1CC)
		ROUND(e, f, g, h, a, b, c, d, schedule20, 0x2DE92C6F)
		ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4A7484AA)
		ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5CB0A9DC)
		ROUND(b, c, d, e, f, g, h, a, schedule23, 0x76F988DA)
		ROUND(a, b, c, d, e, f, g, h, schedule24, 0x983E5152)
		ROUND(h, a, b, c, d, e, f, g, schedule25, 0xA831C66D)
		ROUND(g, h, a, b, c, d, e, f, schedule26, 0xB00327C8)
		ROUND(f, g, h, a, b, c, d, e, schedule27, 0xBF597FC7)
		ROUND(e, f, g, h, a, b, c, d, schedule28, 0xC6E00BF3)
		ROUND(d, e, f, g, h, a, b, c, schedule29, 0xD5A79147)
		ROUND(c, d, e, f, g, h, a, b, schedule30, 0x06CA6351)
		ROUND(b, c, d, e, f, g, h, a, schedule31, 0x14292967)

		ALL_SCHEDULE32()

		ROUND(a, b, c, d, e, f, g, h, schedule0, 0x27B70A85)
		ROUND(h, a, b, c, d, e, f, g, schedule1, 0x2E1B2138)
		ROUND(g, h, a, b, c, d, e, f, schedule2, 0x4D2C6DFC)
		ROUND(f, g, h, a, b, c, d, e, schedule3, 0x53380D13)
		ROUND(e, f, g, h, a, b, c, d, schedule4, 0x650A7354)
		ROUND(d, e, f, g, h, a, b, c, schedule5, 0x766A0ABB)
		ROUND(c, d, e, f, g, h, a, b, schedule6, 0x81C2C92E)
		ROUND(b, c, d, e, f, g, h, a, schedule7, 0x92722C85)
		ROUND(a, b, c, d, e, f, g, h, schedule8, 0xA2BFE8A1)
		ROUND(h, a, b, c, d, e, f, g, schedule9, 0xA81A664B)
		ROUND(g, h, a, b, c, d, e, f, schedule10, 0xC24B8B70)
		ROUND(f, g, h, a, b, c, d, e, schedule11, 0xC76C51A3)
		ROUND(e, f, g, h, a, b, c, d, schedule12, 0xD192E819)
		ROUND(d, e, f, g, h, a, b, c, schedule13, 0xD6990624)
		ROUND(c, d, e, f, g, h, a, b, schedule14, 0xF40E3585)
		ROUND(b, c, d, e, f, g, h, a, schedule15, 0x106AA070)
		ROUND(a, b, c, d, e, f, g, h, schedule16, 0x19A4C116)
		ROUND(h, a, b, c, d, e, f, g, schedule17, 0x1E376C08)
		ROUND(g, h, a, b, c, d, e, f, schedule18, 0x2748774C)
		ROUND(f, g, h, a, b, c, d, e, schedule19, 0x34B0BCB5)
		ROUND(e, f, g, h, a, b, c, d, schedule20, 0x391C0CB3)
		ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4ED8AA4A)
		ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5B9CCA4F)
		ROUND(b, c, d, e, f, g, h, a, schedule23, 0x682E6FF3)
		ROUND(a, b, c, d, e, f, g, h, schedule24, 0x748F82EE)
		ROUND(h, a, b, c, d, e, f, g, schedule25, 0x78A5636F)
		ROUND(g, h, a, b, c, d, e, f, schedule26, 0x84C87814)
		ROUND(f, g, h, a, b, c, d, e, schedule27, 0x8CC70208)
		ROUND(e, f, g, h, a, b, c, d, schedule28, 0x90BEFFFA)
		ROUND(d, e, f, g, h, a, b, c, schedule29, 0xA4506CEB)
		ROUND(c, d, e, f, g, h, a, b, schedule30, 0xBEF9A3F7)
		ROUND(b, c, d, e, f, g, h, a, schedule31, 0xC67178F2)
		
		first_hash0 += a;
		first_hash1 += b;
		first_hash2 += c;
		first_hash3 += d;
		first_hash4 += e;
		first_hash5 += f;
		first_hash6 += g;
		first_hash7 += h;

//----------------------------------------------------- LOOP HASH ------------------------------------------------
		
		hash0=0;
		hash1=0;
		hash2=0;
		hash3=0;
		hash4=0;
		hash5=0;
		hash6=0;
		hash7=0;

		indexW=0;
	//#pragma unroll 1048576
		for(index_generic=0; index_generic < ITERATION_NUMBER/2; index_generic++)
		{
			//Prima parte
			a = UINT32_C(0x6A09E667);
			b = UINT32_C(0xBB67AE85);
			c = UINT32_C(0x3C6EF372);
			d = UINT32_C(0xA54FF53A);
			e = UINT32_C(0x510E527F);
			f = UINT32_C(0x9B05688C);
			g = UINT32_C(0x1F83D9AB);
			h = UINT32_C(0x5BE0CD19);

			schedule0 = hash0;
			schedule1 = hash1;
			schedule2 = hash2;
			schedule3 = hash3;
			schedule4 = hash4;
			schedule5 = hash5;
			schedule6 = hash6;
			schedule7 = hash7;

			schedule8 = first_hash0;
			schedule9 = first_hash1;
			schedule10 = first_hash2;
			schedule11 = first_hash3;
			schedule12 = first_hash4;
			schedule13 = first_hash5;
			schedule14 = first_hash6;
			schedule15 = first_hash7;

			ALL_SCHEDULE_LAST16()

			ROUND(a, b, c, d, e, f, g, h,  schedule0, 0x428A2F98)
			ROUND(h, a, b, c, d, e, f, g,  schedule1, 0x71374491)
			ROUND(g, h, a, b, c, d, e, f,  schedule2, 0xB5C0FBCF)
			ROUND(f, g, h, a, b, c, d, e,  schedule3, 0xE9B5DBA5)
			ROUND(e, f, g, h, a, b, c, d,  schedule4, 0x3956C25B)
			ROUND(d, e, f, g, h, a, b, c,  schedule5, 0x59F111F1)
			ROUND(c, d, e, f, g, h, a, b,  schedule6, 0x923F82A4)
			ROUND(b, c, d, e, f, g, h, a,  schedule7, 0xAB1C5ED5)
			ROUND(a, b, c, d, e, f, g, h,  schedule8, 0xD807AA98)
			ROUND(h, a, b, c, d, e, f, g,  schedule9, 0x12835B01)
			ROUND(g, h, a, b, c, d, e, f, schedule10, 0x243185BE)
			ROUND(f, g, h, a, b, c, d, e, schedule11, 0x550C7DC3)
			ROUND(e, f, g, h, a, b, c, d, schedule12, 0x72BE5D74)
			ROUND(d, e, f, g, h, a, b, c, schedule13, 0x80DEB1FE)
			ROUND(c, d, e, f, g, h, a, b, schedule14, 0x9BDC06A7)
			ROUND(b, c, d, e, f, g, h, a, schedule15, 0xC19BF174)
			ROUND(a, b, c, d, e, f, g, h, schedule16, 0xE49B69C1)
			ROUND(h, a, b, c, d, e, f, g, schedule17, 0xEFBE4786)
			ROUND(g, h, a, b, c, d, e, f, schedule18, 0x0FC19DC6)
			ROUND(f, g, h, a, b, c, d, e, schedule19, 0x240CA1CC)
			ROUND(e, f, g, h, a, b, c, d, schedule20, 0x2DE92C6F)
			ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4A7484AA)
			ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5CB0A9DC)
			ROUND(b, c, d, e, f, g, h, a, schedule23, 0x76F988DA)
			ROUND(a, b, c, d, e, f, g, h, schedule24, 0x983E5152)
			ROUND(h, a, b, c, d, e, f, g, schedule25, 0xA831C66D)
			ROUND(g, h, a, b, c, d, e, f, schedule26, 0xB00327C8)
			ROUND(f, g, h, a, b, c, d, e, schedule27, 0xBF597FC7)
			ROUND(e, f, g, h, a, b, c, d, schedule28, 0xC6E00BF3)
			ROUND(d, e, f, g, h, a, b, c, schedule29, 0xD5A79147)
			ROUND(c, d, e, f, g, h, a, b, schedule30, 0x06CA6351)
			ROUND(b, c, d, e, f, g, h, a, schedule31, 0x14292967)

			ALL_SCHEDULE32()

			ROUND(a, b, c, d, e, f, g, h, schedule0, 0x27B70A85)
			ROUND(h, a, b, c, d, e, f, g, schedule1, 0x2E1B2138)
			ROUND(g, h, a, b, c, d, e, f, schedule2, 0x4D2C6DFC)
			ROUND(f, g, h, a, b, c, d, e, schedule3, 0x53380D13)
			ROUND(e, f, g, h, a, b, c, d, schedule4, 0x650A7354)
			ROUND(d, e, f, g, h, a, b, c, schedule5, 0x766A0ABB)
			ROUND(c, d, e, f, g, h, a, b, schedule6, 0x81C2C92E)
			ROUND(b, c, d, e, f, g, h, a, schedule7, 0x92722C85)
			ROUND(a, b, c, d, e, f, g, h, schedule8, 0xA2BFE8A1)
			ROUND(h, a, b, c, d, e, f, g, schedule9, 0xA81A664B)
			ROUND(g, h, a, b, c, d, e, f, schedule10, 0xC24B8B70)
			ROUND(f, g, h, a, b, c, d, e, schedule11, 0xC76C51A3)
			ROUND(e, f, g, h, a, b, c, d, schedule12, 0xD192E819)
			ROUND(d, e, f, g, h, a, b, c, schedule13, 0xD6990624)
			ROUND(c, d, e, f, g, h, a, b, schedule14, 0xF40E3585)
			ROUND(b, c, d, e, f, g, h, a, schedule15, 0x106AA070)
			ROUND(a, b, c, d, e, f, g, h, schedule16, 0x19A4C116)
			ROUND(h, a, b, c, d, e, f, g, schedule17, 0x1E376C08)
			ROUND(g, h, a, b, c, d, e, f, schedule18, 0x2748774C)
			ROUND(f, g, h, a, b, c, d, e, schedule19, 0x34B0BCB5)
			ROUND(e, f, g, h, a, b, c, d, schedule20, 0x391C0CB3)
			ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4ED8AA4A)
			ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5B9CCA4F)
			ROUND(b, c, d, e, f, g, h, a, schedule23, 0x682E6FF3)
			ROUND(a, b, c, d, e, f, g, h, schedule24, 0x748F82EE)
			ROUND(h, a, b, c, d, e, f, g, schedule25, 0x78A5636F)
			ROUND(g, h, a, b, c, d, e, f, schedule26, 0x84C87814)
			ROUND(f, g, h, a, b, c, d, e, schedule27, 0x8CC70208)
			ROUND(e, f, g, h, a, b, c, d, schedule28, 0x90BEFFFA)
			ROUND(d, e, f, g, h, a, b, c, schedule29, 0xA4506CEB)
			ROUND(c, d, e, f, g, h, a, b, schedule30, 0xBEF9A3F7)
			ROUND(b, c, d, e, f, g, h, a, schedule31, 0xC67178F2)

			hash0 = UINT32_C(0x6A09E667) + a;
			hash1 = UINT32_C(0xBB67AE85) + b;
			hash2 = UINT32_C(0x3C6EF372) + c;
			hash3 = UINT32_C(0xA54FF53A) + d;
			hash4 = UINT32_C(0x510E527F) + e;
			hash5 = UINT32_C(0x9B05688C) + f;
			hash6 = UINT32_C(0x1F83D9AB) + g;
			hash7 = UINT32_C(0x5BE0CD19) + h;

			//Seconda parte
			a = hash0;
			b = hash1;
			c = hash2;
			d = hash3;
			e = hash4;
			f = hash5;
			g = hash6;
			h = hash7;

			//I primi 4 valori dei blocchi W sono sempre uguali
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h,  0, 0x428A2F98, 0)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g,  1, 0x71374491, 0)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f,  2, 0xB5C0FBCF, 0)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e,  3, 0xE9B5DBA5, 0)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d,  4, 0x3956C25B, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c,  5, 0x59F111F1, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b,  6, 0x923F82A4, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a,  7, 0xAB1C5ED5, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h,  8, 0xD807AA98, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g,  9, 0x12835B01, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 10, 0x243185BE, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 11, 0x550C7DC3, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 12, 0x72BE5D74, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 13, 0x80DEB1FE, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 14, 0x9BDC06A7, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 15, 0xC19BF174, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 16, 0xE49B69C1, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 17, 0xEFBE4786, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 18, 0x0FC19DC6, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 19, 0x240CA1CC, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 20, 0x2DE92C6F, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 21, 0x4A7484AA, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 22, 0x5CB0A9DC, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 23, 0x76F988DA, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 24, 0x983E5152, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 25, 0xA831C66D, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 26, 0xB00327C8, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 27, 0xBF597FC7, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 28, 0xC6E00BF3, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 29, 0xD5A79147, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 30, 0x06CA6351, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 31, 0x14292967, indexW)

			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 32, 0x27B70A85, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 33, 0x2E1B2138, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 34, 0x4D2C6DFC, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 35, 0x53380D13, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 36, 0x650A7354, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 37, 0x766A0ABB, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 38, 0x81C2C92E, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 39, 0x92722C85, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 40, 0xA2BFE8A1, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 41, 0xA81A664B, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 42, 0xC24B8B70, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 43, 0xC76C51A3, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 44, 0xD192E819, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 45, 0xD6990624, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 46, 0xF40E3585, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 47, 0x106AA070, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 48, 0x19A4C116, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 49, 0x1E376C08, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 50, 0x2748774C, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 51, 0x34B0BCB5, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 52, 0x391C0CB3, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 53, 0x4ED8AA4A, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 54, 0x5B9CCA4F, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 55, 0x682E6FF3, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 56, 0x748F82EE, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 57, 0x78A5636F, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 58, 0x84C87814, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 59, 0x8CC70208, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 60, 0x90BEFFFA, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 61, 0xA4506CEB, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 62, 0xBEF9A3F7, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 63, 0xC67178F2, indexW)
			
			hash0 += a;
			hash1 += b;
			hash2 += c;
			hash3 += d;
			hash4 += e;
			hash5 += f;
			hash6 += g;
			hash7 += h;

			indexW += SINGLE_BLOCK_W_SIZE;
		}

		for(index_generic=ITERATION_NUMBER/2; index_generic < ITERATION_NUMBER; index_generic++)
		{
			//Prima parte
			a = UINT32_C(0x6A09E667);
			b = UINT32_C(0xBB67AE85);
			c = UINT32_C(0x3C6EF372);
			d = UINT32_C(0xA54FF53A);
			e = UINT32_C(0x510E527F);
			f = UINT32_C(0x9B05688C);
			g = UINT32_C(0x1F83D9AB);
			h = UINT32_C(0x5BE0CD19);

			schedule0 = hash0;
			schedule1 = hash1;
			schedule2 = hash2;
			schedule3 = hash3;
			schedule4 = hash4;
			schedule5 = hash5;
			schedule6 = hash6;
			schedule7 = hash7;

			schedule8 = first_hash0;
			schedule9 = first_hash1;
			schedule10 = first_hash2;
			schedule11 = first_hash3;
			schedule12 = first_hash4;
			schedule13 = first_hash5;
			schedule14 = first_hash6;
			schedule15 = first_hash7;

			ALL_SCHEDULE_LAST16()

			ROUND(a, b, c, d, e, f, g, h,  schedule0, 0x428A2F98)
			ROUND(h, a, b, c, d, e, f, g,  schedule1, 0x71374491)
			ROUND(g, h, a, b, c, d, e, f,  schedule2, 0xB5C0FBCF)
			ROUND(f, g, h, a, b, c, d, e,  schedule3, 0xE9B5DBA5)
			ROUND(e, f, g, h, a, b, c, d,  schedule4, 0x3956C25B)
			ROUND(d, e, f, g, h, a, b, c,  schedule5, 0x59F111F1)
			ROUND(c, d, e, f, g, h, a, b,  schedule6, 0x923F82A4)
			ROUND(b, c, d, e, f, g, h, a,  schedule7, 0xAB1C5ED5)
			ROUND(a, b, c, d, e, f, g, h,  schedule8, 0xD807AA98)
			ROUND(h, a, b, c, d, e, f, g,  schedule9, 0x12835B01)
			ROUND(g, h, a, b, c, d, e, f, schedule10, 0x243185BE)
			ROUND(f, g, h, a, b, c, d, e, schedule11, 0x550C7DC3)
			ROUND(e, f, g, h, a, b, c, d, schedule12, 0x72BE5D74)
			ROUND(d, e, f, g, h, a, b, c, schedule13, 0x80DEB1FE)
			ROUND(c, d, e, f, g, h, a, b, schedule14, 0x9BDC06A7)
			ROUND(b, c, d, e, f, g, h, a, schedule15, 0xC19BF174)
			ROUND(a, b, c, d, e, f, g, h, schedule16, 0xE49B69C1)
			ROUND(h, a, b, c, d, e, f, g, schedule17, 0xEFBE4786)
			ROUND(g, h, a, b, c, d, e, f, schedule18, 0x0FC19DC6)
			ROUND(f, g, h, a, b, c, d, e, schedule19, 0x240CA1CC)
			ROUND(e, f, g, h, a, b, c, d, schedule20, 0x2DE92C6F)
			ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4A7484AA)
			ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5CB0A9DC)
			ROUND(b, c, d, e, f, g, h, a, schedule23, 0x76F988DA)
			ROUND(a, b, c, d, e, f, g, h, schedule24, 0x983E5152)
			ROUND(h, a, b, c, d, e, f, g, schedule25, 0xA831C66D)
			ROUND(g, h, a, b, c, d, e, f, schedule26, 0xB00327C8)
			ROUND(f, g, h, a, b, c, d, e, schedule27, 0xBF597FC7)
			ROUND(e, f, g, h, a, b, c, d, schedule28, 0xC6E00BF3)
			ROUND(d, e, f, g, h, a, b, c, schedule29, 0xD5A79147)
			ROUND(c, d, e, f, g, h, a, b, schedule30, 0x06CA6351)
			ROUND(b, c, d, e, f, g, h, a, schedule31, 0x14292967)

			ALL_SCHEDULE32()

			ROUND(a, b, c, d, e, f, g, h, schedule0, 0x27B70A85)
			ROUND(h, a, b, c, d, e, f, g, schedule1, 0x2E1B2138)
			ROUND(g, h, a, b, c, d, e, f, schedule2, 0x4D2C6DFC)
			ROUND(f, g, h, a, b, c, d, e, schedule3, 0x53380D13)
			ROUND(e, f, g, h, a, b, c, d, schedule4, 0x650A7354)
			ROUND(d, e, f, g, h, a, b, c, schedule5, 0x766A0ABB)
			ROUND(c, d, e, f, g, h, a, b, schedule6, 0x81C2C92E)
			ROUND(b, c, d, e, f, g, h, a, schedule7, 0x92722C85)
			ROUND(a, b, c, d, e, f, g, h, schedule8, 0xA2BFE8A1)
			ROUND(h, a, b, c, d, e, f, g, schedule9, 0xA81A664B)
			ROUND(g, h, a, b, c, d, e, f, schedule10, 0xC24B8B70)
			ROUND(f, g, h, a, b, c, d, e, schedule11, 0xC76C51A3)
			ROUND(e, f, g, h, a, b, c, d, schedule12, 0xD192E819)
			ROUND(d, e, f, g, h, a, b, c, schedule13, 0xD6990624)
			ROUND(c, d, e, f, g, h, a, b, schedule14, 0xF40E3585)
			ROUND(b, c, d, e, f, g, h, a, schedule15, 0x106AA070)
			ROUND(a, b, c, d, e, f, g, h, schedule16, 0x19A4C116)
			ROUND(h, a, b, c, d, e, f, g, schedule17, 0x1E376C08)
			ROUND(g, h, a, b, c, d, e, f, schedule18, 0x2748774C)
			ROUND(f, g, h, a, b, c, d, e, schedule19, 0x34B0BCB5)
			ROUND(e, f, g, h, a, b, c, d, schedule20, 0x391C0CB3)
			ROUND(d, e, f, g, h, a, b, c, schedule21, 0x4ED8AA4A)
			ROUND(c, d, e, f, g, h, a, b, schedule22, 0x5B9CCA4F)
			ROUND(b, c, d, e, f, g, h, a, schedule23, 0x682E6FF3)
			ROUND(a, b, c, d, e, f, g, h, schedule24, 0x748F82EE)
			ROUND(h, a, b, c, d, e, f, g, schedule25, 0x78A5636F)
			ROUND(g, h, a, b, c, d, e, f, schedule26, 0x84C87814)
			ROUND(f, g, h, a, b, c, d, e, schedule27, 0x8CC70208)
			ROUND(e, f, g, h, a, b, c, d, schedule28, 0x90BEFFFA)
			ROUND(d, e, f, g, h, a, b, c, schedule29, 0xA4506CEB)
			ROUND(c, d, e, f, g, h, a, b, schedule30, 0xBEF9A3F7)
			ROUND(b, c, d, e, f, g, h, a, schedule31, 0xC67178F2)

			hash0 = UINT32_C(0x6A09E667) + a;
			hash1 = UINT32_C(0xBB67AE85) + b;
			hash2 = UINT32_C(0x3C6EF372) + c;
			hash3 = UINT32_C(0xA54FF53A) + d;
			hash4 = UINT32_C(0x510E527F) + e;
			hash5 = UINT32_C(0x9B05688C) + f;
			hash6 = UINT32_C(0x1F83D9AB) + g;
			hash7 = UINT32_C(0x5BE0CD19) + h;

			//Seconda parte
			a = hash0;
			b = hash1;
			c = hash2;
			d = hash3;
			e = hash4;
			f = hash5;
			g = hash6;
			h = hash7;

			//I primi 4 valori dei blocchi W sono sempre uguali
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h,  0, 0x428A2F98, 0)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g,  1, 0x71374491, 0)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f,  2, 0xB5C0FBCF, 0)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e,  3, 0xE9B5DBA5, 0)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d,  4, 0x3956C25B, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c,  5, 0x59F111F1, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b,  6, 0x923F82A4, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a,  7, 0xAB1C5ED5, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h,  8, 0xD807AA98, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g,  9, 0x12835B01, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 10, 0x243185BE, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 11, 0x550C7DC3, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 12, 0x72BE5D74, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 13, 0x80DEB1FE, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 14, 0x9BDC06A7, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 15, 0xC19BF174, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 16, 0xE49B69C1, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 17, 0xEFBE4786, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 18, 0x0FC19DC6, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 19, 0x240CA1CC, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 20, 0x2DE92C6F, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 21, 0x4A7484AA, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 22, 0x5CB0A9DC, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 23, 0x76F988DA, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 24, 0x983E5152, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 25, 0xA831C66D, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 26, 0xB00327C8, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 27, 0xBF597FC7, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 28, 0xC6E00BF3, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 29, 0xD5A79147, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 30, 0x06CA6351, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 31, 0x14292967, indexW)

			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 32, 0x27B70A85, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 33, 0x2E1B2138, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 34, 0x4D2C6DFC, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 35, 0x53380D13, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 36, 0x650A7354, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 37, 0x766A0ABB, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 38, 0x81C2C92E, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 39, 0x92722C85, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 40, 0xA2BFE8A1, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 41, 0xA81A664B, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 42, 0xC24B8B70, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 43, 0xC76C51A3, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 44, 0xD192E819, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 45, 0xD6990624, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 46, 0xF40E3585, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 47, 0x106AA070, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 48, 0x19A4C116, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 49, 0x1E376C08, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 50, 0x2748774C, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 51, 0x34B0BCB5, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 52, 0x391C0CB3, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 53, 0x4ED8AA4A, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 54, 0x5B9CCA4F, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 55, 0x682E6FF3, indexW)
			ROUND_SECOND_BLOCK(a, b, c, d, e, f, g, h, 56, 0x748F82EE, indexW)
			ROUND_SECOND_BLOCK(h, a, b, c, d, e, f, g, 57, 0x78A5636F, indexW)
			ROUND_SECOND_BLOCK(g, h, a, b, c, d, e, f, 58, 0x84C87814, indexW)
			ROUND_SECOND_BLOCK(f, g, h, a, b, c, d, e, 59, 0x8CC70208, indexW)
			ROUND_SECOND_BLOCK(e, f, g, h, a, b, c, d, 60, 0x90BEFFFA, indexW)
			ROUND_SECOND_BLOCK(d, e, f, g, h, a, b, c, 61, 0xA4506CEB, indexW)
			ROUND_SECOND_BLOCK(c, d, e, f, g, h, a, b, 62, 0xBEF9A3F7, indexW)
			ROUND_SECOND_BLOCK(b, c, d, e, f, g, h, a, 63, 0xC67178F2, indexW)
			
			hash0 += a;
			hash1 += b;
			hash2 += c;
			hash3 += d;
			hash4 += e;
			hash5 += f;
			hash6 += g;
			hash7 += h;

			indexW += SINGLE_BLOCK_W_SIZE;
		}
//----------------------------------------------------- EXP KEY 256 ------------------------------------------------
		// with Nb=4 and Nk=256 -> Nr=15 so (14+1)*Nb=60 32b-words are needed
		// 48 words
		//----- 1
		// AddRoundKey

		/* REUSE OF SCHEDULE VARIABLES */
		schedule0 = __byte_perm(((uint32_t *)(IV))[0], 0, 0x0123) ^ hash0;
        schedule1 = __byte_perm(((uint32_t *)(IV+4))[0], 0, 0x0123) ^ hash1;
        schedule2 = __byte_perm(((uint32_t *)(IV+8))[0], 0, 0x0123) ^ hash2;
        schedule3 = __byte_perm(((uint32_t *)(IV+12))[0], 0, 0x0123) ^ hash3;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);

		hash0 ^= LOP3LUT_XOR( 
						LOP3LUT_XOR( (TS2[(hash7 >> 24) ] & 0x000000FF), (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000), (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000)), 
							(TS1[(hash7 ) & 0xFF] & 0x0000FF00), 0x01000000
					); //RCON[0];
		hash1 ^= hash0; hash2 ^= hash1; hash3 ^= hash2;

		schedule0 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule4 >> 24], TS1[(schedule5 >> 16) & 0xFF], TS2[(schedule6 >> 8) & 0xFF]) , TS3[schedule7 & 0xFF] , hash0);
		schedule1 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule5 >> 24], TS1[(schedule6 >> 16) & 0xFF], TS2[(schedule7 >> 8) & 0xFF]) , TS3[schedule4 & 0xFF] , hash1);
		schedule2 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule6 >> 24], TS1[(schedule7 >> 16) & 0xFF], TS2[(schedule4 >> 8) & 0xFF]) , TS3[schedule5 & 0xFF] , hash2);
		schedule3 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule7 >> 24], TS1[(schedule4 >> 16) & 0xFF], TS2[(schedule5 >> 8) & 0xFF]) , TS3[schedule6 & 0xFF] , hash3);

		hash4 ^= (TS3[(hash3 >> 24)       ] & 0xFF000000) ^
				  (TS0[(hash3 >> 16) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash3 >>  8) & 0xFF] & 0x0000FF00) ^ 
				  (TS2[(hash3      ) & 0xFF] & 0x000000FF);
		hash5 ^= hash4;
		hash6 ^= hash5;
		hash7 ^= hash6;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);
		
		//----- 2
		hash0 ^= (TS2[(hash7 >> 24)       ] & 0x000000FF) ^
				  (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000) ^
				  (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash7      ) & 0xFF] & 0x0000FF00) ^ 0x02000000; //RCON[1];
		hash1 ^= hash0; hash2 ^= hash1; hash3 ^= hash2;

		schedule0 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule4 >> 24], TS1[(schedule5 >> 16) & 0xFF], TS2[(schedule6 >> 8) & 0xFF]) , TS3[schedule7 & 0xFF] , hash0);
		schedule1 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule5 >> 24], TS1[(schedule6 >> 16) & 0xFF], TS2[(schedule7 >> 8) & 0xFF]) , TS3[schedule4 & 0xFF] , hash1);
		schedule2 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule6 >> 24], TS1[(schedule7 >> 16) & 0xFF], TS2[(schedule4 >> 8) & 0xFF]) , TS3[schedule5 & 0xFF] , hash2);
		schedule3 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule7 >> 24], TS1[(schedule4 >> 16) & 0xFF], TS2[(schedule5 >> 8) & 0xFF]) , TS3[schedule6 & 0xFF] , hash3);

		hash4 ^= (TS3[(hash3 >> 24)       ] & 0xFF000000) ^
				  (TS0[(hash3 >> 16) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash3 >>  8) & 0xFF] & 0x0000FF00) ^ 
				  (TS2[(hash3      ) & 0xFF] & 0x000000FF);
		hash5 ^= hash4;
		hash6 ^= hash5;
		hash7 ^= hash6;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);


		//----- 3
		hash0 ^= (TS2[(hash7 >> 24)       ] & 0x000000FF) ^
				  (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000) ^
				  (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash7      ) & 0xFF] & 0x0000FF00) ^ 0x04000000; //RCON[2];
		hash1 ^= hash0; hash2 ^= hash1; hash3 ^= hash2;

		schedule0 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule4 >> 24], TS1[(schedule5 >> 16) & 0xFF], TS2[(schedule6 >> 8) & 0xFF]) , TS3[schedule7 & 0xFF] , hash0);
		schedule1 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule5 >> 24], TS1[(schedule6 >> 16) & 0xFF], TS2[(schedule7 >> 8) & 0xFF]) , TS3[schedule4 & 0xFF] , hash1);
		schedule2 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule6 >> 24], TS1[(schedule7 >> 16) & 0xFF], TS2[(schedule4 >> 8) & 0xFF]) , TS3[schedule5 & 0xFF] , hash2);
		schedule3 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule7 >> 24], TS1[(schedule4 >> 16) & 0xFF], TS2[(schedule5 >> 8) & 0xFF]) , TS3[schedule6 & 0xFF] , hash3);


		hash4 ^= (TS3[(hash3 >> 24)       ] & 0xFF000000) ^
				  (TS0[(hash3 >> 16) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash3 >>  8) & 0xFF] & 0x0000FF00) ^ 
				  (TS2[(hash3      ) & 0xFF] & 0x000000FF);
		hash5 ^= hash4;
		hash6 ^= hash5;
		hash7 ^= hash6;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);
		

		//----- 4
		hash0 ^= (TS2[(hash7 >> 24)       ] & 0x000000FF) ^
				  (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000) ^
				  (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash7      ) & 0xFF] & 0x0000FF00) ^ 0x08000000; //RCON[3];
		hash1 ^= hash0; hash2 ^= hash1; hash3 ^= hash2;

		schedule0 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule4 >> 24], TS1[(schedule5 >> 16) & 0xFF], TS2[(schedule6 >> 8) & 0xFF]) , TS3[schedule7 & 0xFF] , hash0);
		schedule1 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule5 >> 24], TS1[(schedule6 >> 16) & 0xFF], TS2[(schedule7 >> 8) & 0xFF]) , TS3[schedule4 & 0xFF] , hash1);
		schedule2 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule6 >> 24], TS1[(schedule7 >> 16) & 0xFF], TS2[(schedule4 >> 8) & 0xFF]) , TS3[schedule5 & 0xFF] , hash2);
		schedule3 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule7 >> 24], TS1[(schedule4 >> 16) & 0xFF], TS2[(schedule5 >> 8) & 0xFF]) , TS3[schedule6 & 0xFF] , hash3);
		
		hash4 ^= (TS3[(hash3 >> 24)       ] & 0xFF000000) ^
				  (TS0[(hash3 >> 16) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash3 >>  8) & 0xFF] & 0x0000FF00) ^ 
				  (TS2[(hash3      ) & 0xFF] & 0x000000FF);
		hash5 ^= hash4;
		hash6 ^= hash5;
		hash7 ^= hash6;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);


		//----- 5
		hash0 ^= (TS2[(hash7 >> 24)       ] & 0x000000FF) ^
				  (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000) ^
				  (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash7      ) & 0xFF] & 0x0000FF00) ^ 0x10000000; //RCON[4];
		hash1 ^= hash0; hash2 ^= hash1; hash3 ^= hash2;

		schedule0 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule4 >> 24], TS1[(schedule5 >> 16) & 0xFF], TS2[(schedule6 >> 8) & 0xFF]) , TS3[schedule7 & 0xFF] , hash0);
		schedule1 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule5 >> 24], TS1[(schedule6 >> 16) & 0xFF], TS2[(schedule7 >> 8) & 0xFF]) , TS3[schedule4 & 0xFF] , hash1);
		schedule2 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule6 >> 24], TS1[(schedule7 >> 16) & 0xFF], TS2[(schedule4 >> 8) & 0xFF]) , TS3[schedule5 & 0xFF] , hash2);
		schedule3 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule7 >> 24], TS1[(schedule4 >> 16) & 0xFF], TS2[(schedule5 >> 8) & 0xFF]) , TS3[schedule6 & 0xFF] , hash3);

		hash4 ^= (TS3[(hash3 >> 24)       ] & 0xFF000000) ^
				  (TS0[(hash3 >> 16) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash3 >>  8) & 0xFF] & 0x0000FF00) ^ 
				  (TS2[(hash3      ) & 0xFF] & 0x000000FF);
		hash5 ^= hash4;
		hash6 ^= hash5;
		hash7 ^= hash6;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);


		//----- 6
		hash0 ^= (TS2[(hash7 >> 24)       ] & 0x000000FF) ^
				  (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000) ^
				  (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash7      ) & 0xFF] & 0x0000FF00) ^ 0x20000000; //RCON[5];
		hash1 ^= hash0; hash2 ^= hash1; hash3 ^= hash2;

		schedule0 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule4 >> 24], TS1[(schedule5 >> 16) & 0xFF], TS2[(schedule6 >> 8) & 0xFF]) , TS3[schedule7 & 0xFF] , hash0);
		schedule1 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule5 >> 24], TS1[(schedule6 >> 16) & 0xFF], TS2[(schedule7 >> 8) & 0xFF]) , TS3[schedule4 & 0xFF] , hash1);
		schedule2 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule6 >> 24], TS1[(schedule7 >> 16) & 0xFF], TS2[(schedule4 >> 8) & 0xFF]) , TS3[schedule5 & 0xFF] , hash2);
		schedule3 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule7 >> 24], TS1[(schedule4 >> 16) & 0xFF], TS2[(schedule5 >> 8) & 0xFF]) , TS3[schedule6 & 0xFF] , hash3);

		hash4 ^= (TS3[(hash3 >> 24)       ] & 0xFF000000) ^
				  (TS0[(hash3 >> 16) & 0xFF] & 0x00FF0000) ^
				  (TS1[(hash3 >>  8) & 0xFF] & 0x0000FF00) ^ 
				  (TS2[(hash3      ) & 0xFF] & 0x000000FF);
		hash5 ^= hash4;
		hash6 ^= hash5;
		hash7 ^= hash6;

		schedule4 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule0 >> 24], TS1[(schedule1 >> 16) & 0xFF], TS2[(schedule2 >> 8) & 0xFF]) , TS3[schedule3 & 0xFF] , hash4);
		schedule5 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule1 >> 24], TS1[(schedule2 >> 16) & 0xFF], TS2[(schedule3 >> 8) & 0xFF]) , TS3[schedule0 & 0xFF] , hash5);
		schedule6 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule2 >> 24], TS1[(schedule3 >> 16) & 0xFF], TS2[(schedule0 >> 8) & 0xFF]) , TS3[schedule1 & 0xFF] , hash6);
		schedule7 = LOP3LUT_XOR(LOP3LUT_XOR(TS0[schedule3 >> 24], TS1[(schedule0 >> 16) & 0xFF], TS2[(schedule1 >> 8) & 0xFF]) , TS3[schedule2 & 0xFF] , hash7);

		// last 4 words
		hash0 ^= (TS2[(hash7 >> 24)       ] & 0x000000FF) ^
			  (TS3[(hash7 >> 16) & 0xFF] & 0xFF000000) ^
			  (TS0[(hash7 >>  8) & 0xFF] & 0x00FF0000) ^
			  (TS1[(hash7      ) & 0xFF] & 0x0000FF00) ^ 0x40000000; //RCON[6];
		hash1 ^= hash0;
		hash2 ^= hash1;
		hash3 ^= hash2;

		// NR-th round
		schedule0 = (TS2[(schedule4 >> 24)       ] & 0xFF000000) ^
			 (TS3[(schedule5 >> 16) & 0xFF] & 0x00FF0000) ^
			 (TS0[(schedule6 >>  8) & 0xFF] & 0x0000FF00) ^
			 (TS1[(schedule7      ) & 0xFF] & 0x000000FF) ^ hash0;

		schedule1 = (TS2[(schedule5 >> 24)       ] & 0xFF000000) ^
			 (TS3[(schedule6 >> 16) & 0xFF] & 0x00FF0000) ^
			 (TS0[(schedule7 >>  8) & 0xFF] & 0x0000FF00) ^
			 (TS1[(schedule4      ) & 0xFF] & 0x000000FF) ^ hash1;

		schedule2 = (TS2[(schedule6 >> 24)       ] & 0xFF000000) ^
			 (TS3[(schedule7 >> 16) & 0xFF] & 0x00FF0000) ^
			 (TS0[(schedule4 >>  8) & 0xFF] & 0x0000FF00) ^
			 (TS1[(schedule5      ) & 0xFF] & 0x000000FF) ^ hash2;

		schedule3 = (TS2[(schedule7 >> 24)       ] & 0xFF000000) ^
			 (TS3[(schedule4 >> 16) & 0xFF] & 0x00FF0000) ^
			 (TS0[(schedule5 >>  8) & 0xFF] & 0x0000FF00) ^
			 (TS1[(schedule6      ) & 0xFF] & 0x000000FF) ^ hash3;

		schedule4 = __byte_perm(schedule0, 0, 0x0123);
		schedule5 = __byte_perm(schedule1, 0, 0x0123);
		schedule6 = __byte_perm(schedule2, 0, 0x0123);
		schedule7 = __byte_perm(schedule3, 0, 0x0123);
		
// ------ TEST VMK ------
		if (
			((vmkKey[0] ^ ((uint8_t) schedule4)) == VMK_SIZE) &&
			((vmkKey[1] ^ ((uint8_t) (schedule4 >> 8))) == 0x00) &&
			((vmkKey[8] ^ ((uint8_t) schedule6)) <= 0x05) &&
			((vmkKey[9] ^ ((uint8_t) (schedule6 >> 8))) == 0x20)
		)
		{
			*found = globalIndexPassword;
			break;
		}

// ------ LOOP ------
		globalIndexPassword += (blockDim.x * gridDim.x);
	}

	return;
}
